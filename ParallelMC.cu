
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <string>
#include <random>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

using namespace std;

__global__ void randInit(hiprandState* state){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(1337, idx, 0, &state[idx]);
}

__global__ void monteCarlo(float* forecasts, float mu, float sigma, int iterations, int periods, int dt){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  float eps;

  for (int i = idx; i < iterations; i += stride){
    for (int j = 1; j < periods; j++){
      hiprandState s;
      hiprand_init(1337, idx + j, 0, &s);
      eps = hiprand_normal(&s);
      forecasts[(i * periods) + j] =  forecasts[(i * periods) + (j - 1)] * exp((mu - (pow(sigma, 2) * .5)) * dt + sigma * eps * sqrt((float) dt));
    }
  }

  return;
}

int main(void)
{
  clock_t start, end;
  start = clock();

  float s0;
  float mu;
  float sigma; 
  int startdate;
  int iterations; 
  int increments;
  int dt;
  int periods;

  string fname = "log_returns.csv";
  ifstream returns_file;
  returns_file.open(fname);

  if (returns_file.is_open()) { 
    string line;

    getline(returns_file, line);
    s0 = stof(line);

    getline(returns_file, line);
    mu = stof(line);

    getline(returns_file, line);
    sigma = stof(line);

    getline(returns_file, line);
    startdate = stof(line);

    getline(returns_file, line);
    iterations = stof(line);

    getline(returns_file, line);
    increments = stof(line);

    getline(returns_file, line);
    dt = stof(line);

    periods = increments / dt;

    getline(returns_file, line);
    int return_num = stof(line);
    float* log_returns = new float[return_num];
    
    for (int i = 0; i < return_num; i++)
    {
      getline(returns_file, line);
      log_returns[i] = stof(line);
    }
        
    returns_file.close(); 
  }
  else{
    cout << "Failed to open " << fname;
  }

  int nThreads = 256;
  int nBlocks = (iterations * periods + nThreads - 1) / nThreads;
  
  /*
  nThreads = 1;
  nBlocks = 1;
  */

  float* forecasts;
  hipMallocManaged(&forecasts, iterations * periods * sizeof(float));
  for (int i = 0; i < iterations * periods; i += periods){
    forecasts[i] = s0;
  }

  //curandState* d_state;
  //cudaMalloc(&d_state, nThreads * nBlocks); 
  //randInit<<<nBlocks, nThreads>>>(d_state);
  monteCarlo<<<nBlocks, nThreads>>>(forecasts, mu, sigma, iterations, periods, dt); 

  hipDeviceSynchronize();
  
  ofstream forecasts_file;
  forecasts_file.open("forecasts.csv");
  if (forecasts_file.is_open()){
    forecasts_file << startdate << "," << periods << "\n";
    for (int i = 0; i < iterations; i++){
      for (int j = 0; j < periods; j++){
        forecasts_file << forecasts[(i * periods) + j];
        if (j < periods - 1) {forecasts_file << ",";}
      }
      forecasts_file << endl;
    }
    forecasts_file.close();
  }
  else{
    cout << "Could not open forecast.csv";
  }

  hipFree(forecasts);

  end = clock();
  cout << ((double) (end - start)) / CLOCKS_PER_SEC << "\n";
  
  hipError_t err = hipGetLastError();  // add
  if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; 
  return 0;
}